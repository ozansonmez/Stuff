#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand_kernel.h>
#include <math.h>

extern "C" 
{

	__device__ float rand_expon(float a, hiprandState *state)
	{
		return -log(hiprand_uniform(state))/a;  // x is now random expon by inverse CDF
	} // END rand_expo


	__device__ float psi_calc(float mu_minus, float alpha, float z)
	{
		float psi;
        	// Compute Psi 
                if(mu_minus < alpha){
                	psi = expf( -1/2*pow(alpha-z,2));
                }
                else {
                        psi = expf(  1/2*( pow(mu_minus-alpha,2) - pow(alpha-z,2) ) );
                }
		return psi;
	}

	__global__ void rtruncnorm_kernel(float *vals, int n, 
        	float *mu, float *sigma, 
                float *lo, float *hi,
                int mu_len, int sigma_len,
                int lo_len, int hi_len,
		int rng_seed_a, int rng_seed_b, int rng_seed_c,
                int maxtries)
	{
    		int accepted = 0;
    		int numtries = 0;
		float x;
    		float u;
    		float alpha;
		float psi;
    		float z;
    		float a;
    		float mu_minus;
    		int left_trunc = 0;

	    	// Figure out which thread and block you are in and map these to a single index, "idx"
    		// Usual block/thread indexing...
    		int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    		int blocksize = blockDim.x * blockDim.y * blockDim.z;
    		int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    		int idx = myblock * blocksize + subthread;
    
    		// Check: if index idx < n generate a sample, else in unneeded thread 
    		if(idx<n){

    	    		// Setup the RNG:
	    		hiprandState rng;
	    		hiprand_init(rng_seed_a + idx*rng_seed_b, rng_seed_c, 0, &rng);

	   	    	// Sample the truncated normal
	    		// i.e. pick off mu and sigma corresponding to idx and generate a random sample, x
	    		// if that random sample, x, is in the truncation region, update the return value to x, i.e. vals[idx]=x
	    		// if x is not in the trunc region, try again until you get a sample in the trunc region or if more than maxtries,
	    		// move on to Robert's approx method
	    		while(accepted == 0 && numtries < maxtries){
				numtries++;  // Increment numtries
				x = mu[idx] + sigma[idx]*hiprand_normal(&rng);
				if(x >= lo[idx] && x <= hi[idx]){
					accepted = 1;
					vals[idx] = x;
				}
	    		} 

	    		// Robert's approx method
	    		// We don't want to write both trunc algos for left and right tail truncations, just use 
			// right tail trancation.  If we want to sample from Y~N(mu, sigma, -Inf, b), we transform 
			// first X~N(mu, sigma, -b+2*mu, Inf), use only right truncation, sample from the right 
			// tail to get a X, then transform back Y=2*mu-X to get left truncation sample if needed in Robert.  
		    	if(lo[idx] < mu[idx]) {			// then left truncation
				left_trunc = 1;
				a = -1*hi[idx] + 2*mu[idx];		// flip up to right tail  
	    		}
	    		else {
				a = lo[idx];				// right truncation from a=lo[idx] to infinity
	    		}	
		    	mu_minus = (a-mu[idx])/sigma[idx];

            		// need to find mu_minus but that depends on if lower trunc or upper trunc
	            	alpha = (mu_minus + sqrtf(pow(mu_minus,2) + 4))/2;
			numtries = 1;	//  If couldn't get sample naively, reset and try Robert
	    		while(accepted == 0 && numtries < maxtries){
				
				numtries++;  // Increment numtries

				// Need random expon for Robert no curand_expon function so do inverse CDF
				// F(x) = 1-exp(-alpha*x) --> F^1(x) = -log(U)/alpha where U~Unif[0,1]
				// u = curand_uniform(&rng);
				// x = -1 * log(u)/alpha;  // x is now random expon by inverse CDF 
				z = mu_minus + rand_expon(alpha, &rng);

				// Compute Psi = probability of acceptance
				psi = psi_calc(mu_minus, alpha, z);

				// Check if Random Unif[0,1] < Psi, if so accept, else reject and try again
				u = hiprand_uniform(&rng);
				if (u < psi){
					accepted = 1;	// we now have our vals[idx]
					if (left_trunc == 1){  // since originally left trunc, and flip back to left tail and final transform
						vals[idx] = mu[idx] - sigma[idx]*z;
					}
					else {   // right truncation originally so we're done after final transform
						vals[idx] = mu[idx] + sigma[idx]*z;
					}
				}
	    		}
            		if(accepted == 0){	// Just in case both naive and Roberts fail
	            		vals[idx] = -999;
            		}

    		} // END if (idx<n)
    		return;
	} // END rtruncnorm_kernel
} // END extern "C"
